#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pcl/gpu/containers/device_array.h>
#include <pcl/point_types.h>

// CUDA-Kernel zum Entfernen der Ebenenpunkte
__global__ void removePlanePointsGPU(pcl::PointXYZ* cloud, int* inliers, int numInliers, int totalPoints) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= totalPoints) return;

    for (int i = 0; i < numInliers; i++) {
        if (idx == inliers[i]) {
            cloud[idx].x = NAN;
            cloud[idx].y = NAN;
            cloud[idx].z = NAN;
            return;
        }
    }
}

// Wrapper-Funktion für den CUDA-Kernel
void launchRemovePlanePointsKernel(pcl::gpu::DeviceArray<pcl::PointXYZ>& cloud_device, pcl::gpu::DeviceArray<int>& inliers_gpu) {
    int totalPoints = cloud_device.size();
    int numInliers = inliers_gpu.size();

    int threadsPerBlock = 256;
    int numBlocks = (totalPoints + threadsPerBlock - 1) / threadsPerBlock;

    removePlanePointsGPU<<<numBlocks, threadsPerBlock>>>(
        cloud_device.ptr(), inliers_gpu.ptr(), numInliers, totalPoints
    );

    hipDeviceSynchronize();
}
